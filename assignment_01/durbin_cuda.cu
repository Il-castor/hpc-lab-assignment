#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

#include <chrono>
#include <iostream>

/* Include benchmark-specific header. */
/* Default data type is double, default size is 4000. */
#include "durbin.h"

#ifndef NTHREADS
#define NTHREADS 4
#endif
#define BLOCK_SIZE (32)

#include <hip/hip_runtime.h>
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d (code=%i, name=%s)\n", hipGetErrorString(code), file, line, code, hipGetErrorName(code));
        if (abort)
            exit(code);
    }
}

static void host_init_array(int n, DATA_TYPE *r)
{
  int i;
  #pragma omp parallel for
  for (i = 0; i < n; i++)
  {
    r[i] = (i + 1) / n / 4.0;
  }
}

/* Array initialization. */
__global__ static void init_array(int n, DATA_TYPE* r) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n)
    r[i] = (i + 1) / n / 4.0;
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int n, DATA_TYPE* out) {
  int i;

  for (i = 0; i < n; i++)
  {
    fprintf(stderr, DATA_PRINTF_MODIFIER, out[i]);
    if (i % 20 == 0)
      fprintf(stderr, "\n");
  }
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static void kernel_durbin(int n,
                          DATA_TYPE *r,
                          DATA_TYPE *out)
{
  int i, k;
  DATA_TYPE sum, beta, alpha;
  DATA_TYPE y[2][N];
#pragma scop
  y[0][0] = r[0];
  beta = 1;
  alpha = r[0];


  for (k = 1; k < _PB_N; k++)
  {
    beta = beta - alpha * alpha * beta;

    sum = 0;

    // KERNEL  1
    for (i = 0; i <= k - 1; i++)
      sum += r[k - i - 1] * y[(k - 1) % 2][i];
    // END

    // reduce(sum)
    sum += r[k];
    alpha = -sum * beta;

    // KERNEL 2 (alpha)
    for (i = 0; i <= k - 1; i++)
      y[k % 2][i] = y[(k - 1) % 2][i] + alpha * y[(k - 1) % 2][k - i - 1];
    y[k % 2][k] = alpha;
    // END

    // barrier
  }

  for (i = 0; i < _PB_N; i++)
    out[i] = y[(_PB_N - 1) % 2][i];

}

__global__ static void durbin_k1(int k, DATA_TYPE *r, DATA_TYPE *y, DATA_TYPE  *tmp) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < k)
    tmp[i] = r[k - i - 1] * y[i];
}

/*
 * Applies a sum reduction on an entire warp, the result will be saved in sdata[0]
 * (the others will contain invalid data)
 */
__device__ void warpReduce(volatile DATA_TYPE *sdata, unsigned int tid) {
  if (BLOCK_SIZE >= 64) sdata[tid] += sdata[tid + 32];
  if (BLOCK_SIZE >= 32) sdata[tid] += sdata[tid + 16];
  if (BLOCK_SIZE >= 16) sdata[tid] += sdata[tid + 8];
  if (BLOCK_SIZE >= 8) sdata[tid] += sdata[tid + 4];
  if (BLOCK_SIZE >= 4) sdata[tid] += sdata[tid + 2];
  if (BLOCK_SIZE >= 2) sdata[tid] += sdata[tid + 1];
}

/*
 * every block applies a sum reduction and stores the result in g_odata
 */
__global__ static void blockReduce(DATA_TYPE *g_idata, DATA_TYPE *g_odata, unsigned int n) {
  __shared__ DATA_TYPE sdata[BLOCK_SIZE*2];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(BLOCK_SIZE*2) + tid;
  unsigned int gridSize = BLOCK_SIZE*2*gridDim.x;
  sdata[tid] = 0;
  while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+BLOCK_SIZE]; i += gridSize; }
  __syncthreads();
  if (BLOCK_SIZE >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
  if (BLOCK_SIZE >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
  if (BLOCK_SIZE >= 128) { if (tid < 64 ) { sdata[tid] += sdata[tid + 64];  } __syncthreads(); }
  if (tid < 32) warpReduce(sdata, tid);
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ static void durbin_k2(int k, DATA_TYPE *r, DATA_TYPE *yi, DATA_TYPE *yo, DATA_TYPE alpha) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < k)
    yo[i] = yi[i] + alpha * yi[k - i - 1];
  if (i == k)
    yo[k] = alpha;
}

static long long int hash_array(int n,  DATA_TYPE *out)
{
    long long int hash = 13;
    int i;

    for (i = 0; i < n; i++)
    {
        hash = hash * 37 + *reinterpret_cast<long long int *>(&out[i]);
    }
    return hash;
}

DATA_TYPE reduceCuda(DATA_TYPE *in, DATA_TYPE *tmp, int n) {
  // n -> ceil(n / BLOCK_SIZE) (repeat until we are 1)
  //printf("reduceCuda(%p, %p, %i)\n", (void *) in, (void *) tmp, n);

  while (n != 1) {
    int dimBlock = BLOCK_SIZE;
    int dimGrid = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // Divides "in" in block (by cuda), for every block, reduces it to only 1 number that writes in "out"
    blockReduce<<<dimGrid, dimBlock>>>(in, tmp, n);
    gpuErrchk(hipPeekAtLastError());
    // Since every block writes only 1 element in "out", now we have "dimGrid" elements to reduce
    std::swap(in, tmp);
    n = dimGrid;
  }
  DATA_TYPE out = 0.0;
  gpuErrchk(hipMemcpy(&out, in, sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
  return out;
}


int main(int argc, char **argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  DATA_TYPE *h_r = (DATA_TYPE *) malloc(sizeof(DATA_TYPE) * n);
  DATA_TYPE *h_out = (DATA_TYPE *) malloc(sizeof(DATA_TYPE) * n);
  DATA_TYPE *h_out2 = (DATA_TYPE *) malloc(sizeof(DATA_TYPE) * n);

  /* Initialize array(s). */
  auto begin = std::chrono::high_resolution_clock::now();
  host_init_array(n, h_r);

  /* Run kernel. */
  kernel_durbin(n, h_r, h_out);
  // code to benchmark

  auto end = std::chrono::high_resolution_clock::now();
  std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(end-begin).count() << "s" << std::endl;

  /* Stop and print timer. */

  int dimBlock = BLOCK_SIZE;
  int dimGrid = (n + 1 + BLOCK_SIZE - 1) / BLOCK_SIZE;

  DATA_TYPE *r, *out, *ycurr, *ynext, *red1, *red2;
  gpuErrchk(hipMalloc((void **)&r, N * sizeof(DATA_TYPE)));
  gpuErrchk(hipMalloc((void **)&out, N * sizeof(DATA_TYPE)));
  gpuErrchk(hipMalloc((void **)&ycurr, N * sizeof(DATA_TYPE)));
  gpuErrchk(hipMalloc((void **)&ynext, N * sizeof(DATA_TYPE)));
  gpuErrchk(hipMalloc((void **)&red1, N * sizeof(DATA_TYPE)));
  gpuErrchk(hipMalloc((void **)&red2, N * sizeof(DATA_TYPE)));


  auto begin2 = std::chrono::high_resolution_clock::now();

  init_array<<<dimGrid, dimBlock>>>(n, r);


  DATA_TYPE beta = 1;
  DATA_TYPE alpha = 0;
  gpuErrchk(hipMemcpy(ycurr, r, sizeof(DATA_TYPE), hipMemcpyDeviceToDevice));
  gpuErrchk(hipMemcpy(&alpha, r, sizeof(DATA_TYPE), hipMemcpyDeviceToHost));

  for (unsigned int k = 1; k < _PB_N; k++)
  {
    beta = beta - alpha * alpha * beta;

    durbin_k1<<<dimGrid, dimBlock>>>(k, r, ycurr, red1);
    gpuErrchk(hipPeekAtLastError());
    DATA_TYPE sum = reduceCuda(red1, red2, k);

    DATA_TYPE rk;
    gpuErrchk(hipMemcpy(&rk, &r[k], sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
    sum += rk;
    alpha = -sum * beta;

    durbin_k2<<<dimGrid, dimBlock>>>(k, r, ycurr, ynext, alpha);
    gpuErrchk(hipPeekAtLastError());
    std::swap(ycurr, ynext);
  }

  gpuErrchk(hipMemcpy(h_out2, ycurr, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost));


  auto end2 = std::chrono::high_resolution_clock::now();
  std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(end2-begin2).count() << "s" << std::endl;

  gpuErrchk(hipFree(r));
  gpuErrchk(hipFree(out));
  gpuErrchk(hipFree(ycurr));
  gpuErrchk(hipFree(ynext));
  gpuErrchk(hipFree(red1));
  gpuErrchk(hipFree(red2));

  if (argc > 42 && ! strcmp(argv[0], "")) print_array(n, h_out2);
#ifdef PRINT_HASH
  long long int h1 = hash_array(n, h_out);
  long long int h2 = hash_array(n, h_out2);
  printf("%lli\n%lli\n", h1, h2);

  if (h1 != h2) {
    printf("\n\n\n-----------HASHS DIFFER-----------\n");
  }
  double diff = 0.0;
  for (int i = 0; i < n; i++) {
    double d = h_out[i] - h_out2[i];
    diff += d*d;
    std::cout << "INDEX " << i << " " <<  h_out[i] << " " <<  h_out2[i] << " diff:" << (h_out2[i] - h_out[i]) << std::endl;
  }
  std::cout << "NORM DIFFERENCE: " << diff << std::endl;
#endif
#ifdef PRINT_ARRAY
  print_array(n, h_out2);
#endif

  /* Be clean. */
  free(h_r);
  free(h_out);
  free(h_out2);

  return 0;
}
