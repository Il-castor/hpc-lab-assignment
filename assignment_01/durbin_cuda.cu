#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

#include <chrono>
#include <iostream>

/* Include benchmark-specific header. */
/* Default data type is double, default size is 4000. */
#include "durbin.h"

#ifndef NTHREADS
#define NTHREADS 4
#endif
#define BLOCK_SIZE (64)

#define SUM_BLOCK_COUNT (4)

#include <hip/hip_runtime.h>
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d (code=%i, name=%s)\n", hipGetErrorString(code), file, line, code, hipGetErrorName(code));
        if (abort)
            exit(code);
    }
}

static void host_init_array(int n, DATA_TYPE *r)
{
  int i;
  #pragma omp parallel for
  for (i = 0; i < n; i++)
  {
    r[i] = (i + 1) / n / 4.0;
  }
}

/* Array initialization. */
__global__ static void init_array(int n, DATA_TYPE* r) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n)
    r[i] = (i + 1) / n / 4.0;
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int n, DATA_TYPE* out) {
  int i;

  for (i = 0; i < n; i++)
  {
    fprintf(stderr, DATA_PRINTF_MODIFIER, out[i]);
    if (i % 20 == 0)
      fprintf(stderr, "\n");
  }
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static void kernel_durbin(int n,
                          DATA_TYPE *r,
                          DATA_TYPE *out)
{
  int i, k;
  DATA_TYPE sum, beta, alpha;
  DATA_TYPE y[2][N];
#pragma scop
  y[0][0] = r[0];
  beta = 1;
  alpha = r[0];


  for (k = 1; k < _PB_N; k++)
  {

    beta = beta - alpha * alpha * beta;

    sum = 0;

    // KERNEL  1
    for (i = 0; i <= k - 1; i++)
      sum += r[k - i - 1] * y[(k - 1) % 2][i];
    // END

    // reduce(sum)
    sum += r[k];
    alpha = -sum * beta;

    // KERNEL 2 (alpha)
    for (i = 0; i <= k - 1; i++)
      y[k % 2][i] = y[(k - 1) % 2][i] + alpha * y[(k - 1) % 2][k - i - 1];
    y[k % 2][k] = alpha;
    // END

    // barrier
  }

  for (i = 0; i < _PB_N; i++)
    out[i] = y[(_PB_N - 1) % 2][i];

}

__device__ DATA_TYPE d_alpha, d_beta;

/*
 * Applies a sum reduction on an entire warp, the result will be saved in sdata[0]
 * (the others will contain invalid data)
 */
__device__ void warpReduce(volatile DATA_TYPE *sdata, unsigned int tid) {
  if (BLOCK_SIZE >= 64) sdata[tid] += sdata[tid + 32];
  if (BLOCK_SIZE >= 32) sdata[tid] += sdata[tid + 16];
  if (BLOCK_SIZE >= 16) sdata[tid] += sdata[tid + 8];
  if (BLOCK_SIZE >= 8) sdata[tid] += sdata[tid + 4];
  if (BLOCK_SIZE >= 4) sdata[tid] += sdata[tid + 2];
  if (BLOCK_SIZE >= 2) sdata[tid] += sdata[tid + 1];
}

/*
 * every block applies a sum reduction and stores the result in g_odata
 */
__global__ static void durbin_k1(unsigned int k, DATA_TYPE *r, DATA_TYPE *y, DATA_TYPE *g_odata) {
  __shared__ DATA_TYPE sdata[BLOCK_SIZE*2];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(BLOCK_SIZE*2) + tid;
  unsigned int gridSize = BLOCK_SIZE*gridDim.x;

  unsigned int workPerThread = (k + gridSize - 1) / gridSize;

  d_beta = d_beta - d_alpha * d_alpha * d_beta;

  DATA_TYPE sum = 0;
  while(workPerThread-- != 0) {
    if (i < k) sum += r[k - i - 1] * y[i];
    i += gridSize;
  }
  sdata[tid] = sum;
  __syncthreads();
  if (BLOCK_SIZE >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
  if (BLOCK_SIZE >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
  if (BLOCK_SIZE >= 128) { if (tid < 64 ) { sdata[tid] += sdata[tid + 64];  } __syncthreads(); }
  if (tid < 32) warpReduce(sdata, tid);

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


__global__ static void durbin_k2(int k, DATA_TYPE *part_sum, DATA_TYPE *r, DATA_TYPE *yi, DATA_TYPE *yo) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;

  __shared__ DATA_TYPE psum[SUM_BLOCK_COUNT];
  if (tid < SUM_BLOCK_COUNT) psum[tid] = part_sum[tid];
  __syncthreads();
  if (i == k) {
    DATA_TYPE sum = 0.0;;
    for (int j = 0; j < SUM_BLOCK_COUNT; j++) sum += psum[j];

    sum += r[k];

    d_alpha = -sum * d_beta;
  }
  __syncthreads();

  DATA_TYPE a = d_alpha;
  if (i < k)
    yo[i] = yi[i] + a * yi[k - i - 1];
  if (i == k)
    yo[k] = a;
}

static long long int hash_array(int n,  DATA_TYPE *out)
{
    long long int hash = 13;
    int i;

    for (i = 0; i < n; i++)
    {
        hash = hash * 37 + *reinterpret_cast<long long int *>(&out[i]);
    }
    return hash;
}


int main(int argc, char **argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  DATA_TYPE *h_r = (DATA_TYPE *) malloc(sizeof(DATA_TYPE) * n);
  DATA_TYPE *h_out = (DATA_TYPE *) malloc(sizeof(DATA_TYPE) * n);
  DATA_TYPE *h_out2 = (DATA_TYPE *) malloc(sizeof(DATA_TYPE) * n);

  /* Initialize array(s). */
  auto begin = std::chrono::high_resolution_clock::now();
  host_init_array(n, h_r);

  /* Run kernel. */
  kernel_durbin(n, h_r, h_out);
  // code to benchmark

  auto end = std::chrono::high_resolution_clock::now();
  std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(end-begin).count() << "s" << std::endl;

  /* Stop and print timer. */

  int dimBlock = BLOCK_SIZE;
  int dimGrid = (n + 1 + BLOCK_SIZE - 1) / BLOCK_SIZE;

  DATA_TYPE *r, *out, *ycurr, *ynext, *part_sum;
  gpuErrchk(hipMalloc((void **)&r, N * sizeof(DATA_TYPE)));
  gpuErrchk(hipMalloc((void **)&out, N * sizeof(DATA_TYPE)));
  gpuErrchk(hipMalloc((void **)&ycurr, N * sizeof(DATA_TYPE)));
  gpuErrchk(hipMalloc((void **)&ynext, N * sizeof(DATA_TYPE)));
  gpuErrchk(hipMalloc((void **)&part_sum, SUM_BLOCK_COUNT * sizeof(DATA_TYPE)));


  auto begin2 = std::chrono::high_resolution_clock::now();

  init_array<<<dimGrid, dimBlock>>>(n, r);

  DATA_TYPE beta = 1;
  DATA_TYPE alpha = 0;
  gpuErrchk(hipMemcpy(ycurr, r, sizeof(DATA_TYPE), hipMemcpyDeviceToDevice));
  gpuErrchk(hipMemcpy(&alpha, r, sizeof(DATA_TYPE), hipMemcpyDeviceToHost));

  hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &alpha, sizeof(DATA_TYPE));
  hipMemcpyToSymbol(HIP_SYMBOL(d_beta), &beta, sizeof(DATA_TYPE));
  for (unsigned int k = 1; k < _PB_N; k++)
  {
    durbin_k1<<<SUM_BLOCK_COUNT, dimBlock>>>(k, r, ycurr, part_sum);
    gpuErrchk(hipPeekAtLastError());
    int dimGrid2 = (k + 1 + BLOCK_SIZE - 1) / BLOCK_SIZE;
    durbin_k2<<<dimGrid2, dimBlock>>>(k, part_sum, r, ycurr, ynext);
    gpuErrchk(hipPeekAtLastError());
    std::swap(ycurr, ynext);
  }

  gpuErrchk(hipMemcpy(h_out2, ycurr, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost));

  auto end2 = std::chrono::high_resolution_clock::now();
  std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(end2-begin2).count() << "s" << std::endl;

  gpuErrchk(hipFree(r));
  gpuErrchk(hipFree(out));
  gpuErrchk(hipFree(ycurr));
  gpuErrchk(hipFree(ynext));
  gpuErrchk(hipFree(part_sum));

  if (argc > 42 && ! strcmp(argv[0], "")) print_array(n, h_out2);
#ifdef PRINT_HASH
  long long int h1 = hash_array(n, h_out);
  long long int h2 = hash_array(n, h_out2);
  printf("%lli\n%lli\n", h1, h2);

  if (h1 != h2) {
    printf("\n\n\n-----------HASHS DIFFER-----------\n");
  }
  double diff = 0.0;
  for (int i = 0; i < n; i++) {
    double d = h_out[i] - h_out2[i];
    diff += d*d;
    if (h_out[i] != h_out2[i])
    std::cout << "INDEX " << i << " " <<  h_out[i] << " " <<  h_out2[i] << " diff:" << (h_out2[i] - h_out[i]) << std::endl;
  }
  std::cout << "NORM DIFFERENCE: " << diff << std::endl;
#endif
#ifdef PRINT_ARRAY
  print_array(n, h_out2);
#endif

  /* Be clean. */
  free(h_r);
  free(h_out);
  free(h_out2);

  return 0;
}
