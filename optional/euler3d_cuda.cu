// Copyright 2009, Andrew Corrigan, acorriga@gmu.edu
// This code is from the AIAA-2009-4001 paper

#include <iostream>
#include <fstream>
#include <cmath>
#include <omp.h>

#ifndef block_length
#define block_length 32
#endif

/*
 * Options
 *
 */
#define GAMMA 1.4
#define iterations 2000

#define NDIM 3
#define NNB 4

#define RK 3 // 3rd order RK
#define ff_mach 1.2
#define deg_angle_of_attack 0.0f

/*
 * not options
 */
#define VAR_DENSITY 0
#define VAR_MOMENTUM 1
#define VAR_DENSITY_ENERGY (VAR_MOMENTUM + NDIM)
#define NVAR (VAR_DENSITY_ENERGY + 1)

#ifdef restrict
#define __restrict restrict
#else
#define __restrict
#endif


#ifndef BLOCK_SIZE
#define BLOCK_SIZE (32)
#endif

#include <hip/hip_runtime.h>
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

/*
 * Generic functions
 */
template <typename T>
T *alloc(int N)
{
	return new T[N];
}

template <typename T>
void dealloc(T *array)
{
	delete[] array;
}

template <typename T>
void copy(T *dst, T *src, int N)
{
	for (int i = 0; i < N; i++)
	{
		dst[i] = src[i];
	}
}

void dump(float *variables, int nel, int nelr)
{
	{
		std::ofstream file("density");
		file << nel << " " << nelr << std::endl;
		for (int i = 0; i < nel; i++)
			file << variables[i + VAR_DENSITY * nelr] << std::endl;
	}

	{
		std::ofstream file("momentum");
		file << nel << " " << nelr << std::endl;
		for (int i = 0; i < nel; i++)
		{
			for (int j = 0; j != NDIM; j++)
				file << variables[i + (VAR_MOMENTUM + j) * nelr] << " ";
			file << std::endl;
		}
	}

	{
		std::ofstream file("density_energy");
		file << nel << " " << nelr << std::endl;
		for (int i = 0; i < nel; i++)
			file << variables[i + VAR_DENSITY_ENERGY * nelr] << std::endl;
	}
}

/*Creazione costanti che utilizza cuda per fare i diversi calcoli*/
__constant__ float ff_variable[NVAR];
__constant__ float3 ff_flux_contribution_momentum_x;
__constant__ float3 ff_flux_contribution_momentum_y;
__constant__ float3 ff_flux_contribution_momentum_z;
__constant__ float3 ff_flux_contribution_density_energy;


//funzione per inizializzare le variabili
__global__ void cuda_initialize_variables(int nelr, float* variables)
{
	/* Original code of this function
	for (int i = 0; i < nelr; i++)
	{
		for (int j = 0; j < NVAR; j++)
			variables[i + j * nelr] = ff_variable[j];
	}
	*/
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	for (int j = 0; j < NVAR; j++)
		variables[i + j * nelr] = ff_variable[j];
}

void initialize_variables(int nelr, float *variables)
{
	dim3 dimGrid(nelr / BLOCK_SIZE),    dimBlock(BLOCK_SIZE);
	cuda_initialize_variables<<<dimGrid, dimBlock>>>(nelr, variables);
	gpuErrchk(hipPeekAtLastError());

}
// questa funzione ?? eseguita sull'host ed ?? chiamata sia dal device che dall'host
__device__ __host__ inline void compute_flux_contribution(float &density, float3 &momentum, float &density_energy, float &pressure, float3 &velocity, float3 &fc_momentum_x, float3 &fc_momentum_y, float3 &fc_momentum_z, float3 &fc_density_energy)
{
	fc_momentum_x.x = velocity.x * momentum.x + pressure;
	fc_momentum_x.y = velocity.x * momentum.y;
	fc_momentum_x.z = velocity.x * momentum.z;

	fc_momentum_y.x = fc_momentum_x.y;
	fc_momentum_y.y = velocity.y * momentum.y + pressure;
	fc_momentum_y.z = velocity.y * momentum.z;

	fc_momentum_z.x = fc_momentum_x.z;
	fc_momentum_z.y = fc_momentum_y.z;
	fc_momentum_z.z = velocity.z * momentum.z + pressure;

	float de_p = density_energy + pressure;
	fc_density_energy.x = velocity.x * de_p;
	fc_density_energy.y = velocity.y * de_p;
	fc_density_energy.z = velocity.z * de_p;
}

// funzione eseguita dal device e chiamabile solamente dal device
__device__ inline void compute_velocity(float &density, float3 &momentum, float3 &velocity)
{
	velocity.x = momentum.x / density;
	velocity.y = momentum.y / density;
	velocity.z = momentum.z / density;
}
// funzione eseguita dal device e chiamabile solamente dal device
__device__ inline float compute_speed_sqd(float3 &velocity)
{
	return velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z;
}
// funzione eseguita dal device e chiamabile solamente dal device
__device__ inline float compute_pressure(float &density, float &density_energy, float &speed_sqd)
{
	return (float(GAMMA) - float(1.0f)) * (density_energy - float(0.5f) * density * speed_sqd);
}
// funzione eseguita dal device e chiamabile solamente dal device
__device__ inline float compute_speed_of_sound(float &density, float &pressure)
{
	return std::sqrt(float(GAMMA) * pressure / density);
}
//funzione eseguita dal device e chiamabile solo dall'host
__global__ void compute_step_factor(int nelr, float *__restrict variables, float *areas, float *__restrict step_factors)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//for (int blk = 0; blk < nelr / block_length; ++blk)
	//{
		//int b_start = blk * block_length;
		//int b_end = (blk + 1) * block_length > nelr ? nelr : (blk + 1) * block_length;
		//for (int i = b_start; i < b_end; i++)
		//{
			float density = variables[i + VAR_DENSITY * nelr];

			float3 momentum;
			momentum.x = variables[i + (VAR_MOMENTUM + 0) * nelr];
			momentum.y = variables[i + (VAR_MOMENTUM + 1) * nelr];
			momentum.z = variables[i + (VAR_MOMENTUM + 2) * nelr];

			float density_energy = variables[i + VAR_DENSITY_ENERGY * nelr];
			float3 velocity;
			compute_velocity(density, momentum, velocity);
			float speed_sqd = compute_speed_sqd(velocity);
			float pressure = compute_pressure(density, density_energy, speed_sqd);
			float speed_of_sound = compute_speed_of_sound(density, pressure);

			// dt = float(0.5f) * std::sqrt(areas[i]) /  (||v|| + c).... but when we do time stepping, this later would need to be divided by the area, so we just do it all at once
			step_factors[i] = float(0.5f) / (std::sqrt(areas[i]) * (std::sqrt(speed_sqd) + speed_of_sound));
		//}
	//}
}

/*
 *
 *
 */
//funzione eseguita dal device e chiamabile solo dall'host
__global__ void compute_flux(int nelr, int *elements_surrounding_elements, float *normals, float *variables, float *fluxes)
{
	const float smoothing_coefficient = float(0.2f);
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j, nb;
	// for (int blk = 0; blk < nelr / block_length; ++blk)
	// {
	// 	int b_start = blk * block_length;
	// 	int b_end = (blk + 1) * block_length > nelr ? nelr : (blk + 1) * block_length;
	// 	for (int i = b_start; i < b_end; ++i)
	// 	{
			float density_i = variables[i + VAR_DENSITY * nelr];
			float3 momentum_i;
			momentum_i.x = variables[i + (VAR_MOMENTUM + 0) * nelr];
			momentum_i.y = variables[i + (VAR_MOMENTUM + 1) * nelr];
			momentum_i.z = variables[i + (VAR_MOMENTUM + 2) * nelr];

			float density_energy_i = variables[i + VAR_DENSITY_ENERGY * nelr];

			float3 velocity_i;
			compute_velocity(density_i, momentum_i, velocity_i);
			float speed_sqd_i = compute_speed_sqd(velocity_i);
			float speed_i = std::sqrt(speed_sqd_i);
			float pressure_i = compute_pressure(density_i, density_energy_i, speed_sqd_i);
			float speed_of_sound_i = compute_speed_of_sound(density_i, pressure_i);
			float3 flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z;
			float3 flux_contribution_i_density_energy;
			compute_flux_contribution(density_i, momentum_i, density_energy_i, pressure_i, velocity_i, flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z, flux_contribution_i_density_energy);

			float flux_i_density = float(0.0f);
			float3 flux_i_momentum;
			flux_i_momentum.x = float(0.0f);
			flux_i_momentum.y = float(0.0f);
			flux_i_momentum.z = float(0.0f);
			float flux_i_density_energy = float(0.0f);

			float3 velocity_nb;
			float density_nb, density_energy_nb;
			float3 momentum_nb;
			float3 flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z;
			float3 flux_contribution_nb_density_energy;
			float speed_sqd_nb, speed_of_sound_nb, pressure_nb;
		//FORSE SI PUO' PARALLELIZZARE ANCHE QUESTO CON CUDA
#pragma unroll
			for (j = 0; j < NNB; j++)
			{
				float3 normal;
				float normal_len;
				float factor;

				nb = elements_surrounding_elements[i + j * nelr];
				normal.x = normals[i + (j + 0 * NNB) * nelr];
				normal.y = normals[i + (j + 1 * NNB) * nelr];
				normal.z = normals[i + (j + 2 * NNB) * nelr];
				normal_len = std::sqrt(normal.x * normal.x + normal.y * normal.y + normal.z * normal.z);

				if (nb >= 0) // a legitimate neighbor
				{
					density_nb = variables[nb + VAR_DENSITY * nelr];
					momentum_nb.x = variables[nb + (VAR_MOMENTUM + 0) * nelr];
					momentum_nb.y = variables[nb + (VAR_MOMENTUM + 1) * nelr];
					momentum_nb.z = variables[nb + (VAR_MOMENTUM + 2) * nelr];
					density_energy_nb = variables[nb + VAR_DENSITY_ENERGY * nelr];
					compute_velocity(density_nb, momentum_nb, velocity_nb);
					speed_sqd_nb = compute_speed_sqd(velocity_nb);
					pressure_nb = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
					speed_of_sound_nb = compute_speed_of_sound(density_nb, pressure_nb);
					compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);

					// artificial viscosity
					factor = -normal_len * smoothing_coefficient * float(0.5f) * (speed_i + std::sqrt(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
					flux_i_density += factor * (density_i - density_nb);
					flux_i_density_energy += factor * (density_energy_i - density_energy_nb);
					flux_i_momentum.x += factor * (momentum_i.x - momentum_nb.x);
					flux_i_momentum.y += factor * (momentum_i.y - momentum_nb.y);
					flux_i_momentum.z += factor * (momentum_i.z - momentum_nb.z);

					// accumulate cell-centered fluxes
					factor = float(0.5f) * normal.x;
					flux_i_density += factor * (momentum_nb.x + momentum_i.x);
					flux_i_density_energy += factor * (flux_contribution_nb_density_energy.x + flux_contribution_i_density_energy.x);
					flux_i_momentum.x += factor * (flux_contribution_nb_momentum_x.x + flux_contribution_i_momentum_x.x);
					flux_i_momentum.y += factor * (flux_contribution_nb_momentum_y.x + flux_contribution_i_momentum_y.x);
					flux_i_momentum.z += factor * (flux_contribution_nb_momentum_z.x + flux_contribution_i_momentum_z.x);

					factor = float(0.5f) * normal.y;
					flux_i_density += factor * (momentum_nb.y + momentum_i.y);
					flux_i_density_energy += factor * (flux_contribution_nb_density_energy.y + flux_contribution_i_density_energy.y);
					flux_i_momentum.x += factor * (flux_contribution_nb_momentum_x.y + flux_contribution_i_momentum_x.y);
					flux_i_momentum.y += factor * (flux_contribution_nb_momentum_y.y + flux_contribution_i_momentum_y.y);
					flux_i_momentum.z += factor * (flux_contribution_nb_momentum_z.y + flux_contribution_i_momentum_z.y);

					factor = float(0.5f) * normal.z;
					flux_i_density += factor * (momentum_nb.z + momentum_i.z);
					flux_i_density_energy += factor * (flux_contribution_nb_density_energy.z + flux_contribution_i_density_energy.z);
					flux_i_momentum.x += factor * (flux_contribution_nb_momentum_x.z + flux_contribution_i_momentum_x.z);
					flux_i_momentum.y += factor * (flux_contribution_nb_momentum_y.z + flux_contribution_i_momentum_y.z);
					flux_i_momentum.z += factor * (flux_contribution_nb_momentum_z.z + flux_contribution_i_momentum_z.z);
				}
				else if (nb == -1) // a wing boundary
				{
					flux_i_momentum.x += normal.x * pressure_i;
					flux_i_momentum.y += normal.y * pressure_i;
					flux_i_momentum.z += normal.z * pressure_i;
				}
				else if (nb == -2) // a far field boundary
				{
					factor = float(0.5f) * normal.x;
					flux_i_density += factor * (ff_variable[VAR_MOMENTUM + 0] + momentum_i.x);
					flux_i_density_energy += factor * (ff_flux_contribution_density_energy.x + flux_contribution_i_density_energy.x);
					flux_i_momentum.x += factor * (ff_flux_contribution_momentum_x.x + flux_contribution_i_momentum_x.x);
					flux_i_momentum.y += factor * (ff_flux_contribution_momentum_y.x + flux_contribution_i_momentum_y.x);
					flux_i_momentum.z += factor * (ff_flux_contribution_momentum_z.x + flux_contribution_i_momentum_z.x);

					factor = float(0.5f) * normal.y;
					flux_i_density += factor * (ff_variable[VAR_MOMENTUM + 1] + momentum_i.y);
					flux_i_density_energy += factor * (ff_flux_contribution_density_energy.y + flux_contribution_i_density_energy.y);
					flux_i_momentum.x += factor * (ff_flux_contribution_momentum_x.y + flux_contribution_i_momentum_x.y);
					flux_i_momentum.y += factor * (ff_flux_contribution_momentum_y.y + flux_contribution_i_momentum_y.y);
					flux_i_momentum.z += factor * (ff_flux_contribution_momentum_z.y + flux_contribution_i_momentum_z.y);

					factor = float(0.5f) * normal.z;
					flux_i_density += factor * (ff_variable[VAR_MOMENTUM + 2] + momentum_i.z);
					flux_i_density_energy += factor * (ff_flux_contribution_density_energy.z + flux_contribution_i_density_energy.z);
					flux_i_momentum.x += factor * (ff_flux_contribution_momentum_x.z + flux_contribution_i_momentum_x.z);
					flux_i_momentum.y += factor * (ff_flux_contribution_momentum_y.z + flux_contribution_i_momentum_y.z);
					flux_i_momentum.z += factor * (ff_flux_contribution_momentum_z.z + flux_contribution_i_momentum_z.z);
				}
			}
			fluxes[i + VAR_DENSITY * nelr] = flux_i_density;
			fluxes[i + (VAR_MOMENTUM + 0) * nelr] = flux_i_momentum.x;
			fluxes[i + (VAR_MOMENTUM + 1) * nelr] = flux_i_momentum.y;
			fluxes[i + (VAR_MOMENTUM + 2) * nelr] = flux_i_momentum.z;
			fluxes[i + VAR_DENSITY_ENERGY * nelr] = flux_i_density_energy;
		//}
	//}
}

__global__ void time_step(int j, int nelr, float *old_variables, float *variables, float *step_factors, float *fluxes)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	// for (int blk = 0; blk < nelr / block_length; ++blk)
	// {
	// 	int b_start = blk * block_length;
	// 	int b_end = (blk + 1) * block_length > nelr ? nelr : (blk + 1) * block_length;
	// 	for (int i = b_start; i < b_end; ++i)
	// 	{
			float factor = step_factors[i] / float(RK + 1 - j);

			variables[i + VAR_DENSITY * nelr] = old_variables[i + VAR_DENSITY * nelr] + factor * fluxes[i + VAR_DENSITY * nelr];
			variables[i + (VAR_MOMENTUM + 0) * nelr] = old_variables[i + (VAR_MOMENTUM + 0) * nelr] + factor * fluxes[i + (VAR_MOMENTUM + 0) * nelr];
			variables[i + (VAR_MOMENTUM + 1) * nelr] = old_variables[i + (VAR_MOMENTUM + 1) * nelr] + factor * fluxes[i + (VAR_MOMENTUM + 1) * nelr];
			variables[i + (VAR_MOMENTUM + 2) * nelr] = old_variables[i + (VAR_MOMENTUM + 2) * nelr] + factor * fluxes[i + (VAR_MOMENTUM + 2) * nelr];
			variables[i + VAR_DENSITY_ENERGY * nelr] = old_variables[i + VAR_DENSITY_ENERGY * nelr] + factor * fluxes[i + VAR_DENSITY_ENERGY * nelr];
		//}
	//}
}
/*
 * Main function
 */
int main(int argc, char **argv)
{
	if (argc < 2)
	{
		std::cout << "specify data file name" << std::endl;
		return 0;
	}
	const char *data_file_name = argv[1];

	float h_ff_variable[NVAR];
	float3 h_ff_flux_contribution_momentum_x, h_ff_flux_contribution_momentum_y, h_ff_flux_contribution_momentum_z, h_ff_flux_contribution_density_energy;

	// set far field conditions
	{
		const float angle_of_attack = float(3.1415926535897931 / 180.0f) * float(deg_angle_of_attack);

		h_ff_variable[VAR_DENSITY] = float(1.4);

		float ff_pressure = float(1.0f);
		float ff_speed_of_sound = sqrt(GAMMA * ff_pressure / h_ff_variable[VAR_DENSITY]);
		float ff_speed = float(ff_mach) * ff_speed_of_sound;

		float3 ff_velocity;
		ff_velocity.x = ff_speed * float(cos((float)angle_of_attack));
		ff_velocity.y = ff_speed * float(sin((float)angle_of_attack));
		ff_velocity.z = 0.0f;

		h_ff_variable[VAR_MOMENTUM + 0] = h_ff_variable[VAR_DENSITY] * ff_velocity.x;
		h_ff_variable[VAR_MOMENTUM + 1] = h_ff_variable[VAR_DENSITY] * ff_velocity.y;
		h_ff_variable[VAR_MOMENTUM + 2] = h_ff_variable[VAR_DENSITY] * ff_velocity.z;

		h_ff_variable[VAR_DENSITY_ENERGY] = h_ff_variable[VAR_DENSITY] * (float(0.5f) * (ff_speed * ff_speed)) + (ff_pressure / float(GAMMA - 1.0f));

		float3 h_ff_momentum;
		h_ff_momentum.x = *(h_ff_variable + VAR_MOMENTUM + 0);
		h_ff_momentum.y = *(h_ff_variable + VAR_MOMENTUM + 1);
		h_ff_momentum.z = *(h_ff_variable + VAR_MOMENTUM + 2);
		compute_flux_contribution(h_ff_variable[VAR_DENSITY], h_ff_momentum, h_ff_variable[VAR_DENSITY_ENERGY], ff_pressure, ff_velocity, h_ff_flux_contribution_momentum_x, h_ff_flux_contribution_momentum_y, h_ff_flux_contribution_momentum_z, h_ff_flux_contribution_density_energy);

		//alloco memoria per la GPU
		// copy far field conditions to the gpu
		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ff_variable),          h_ff_variable,          NVAR*sizeof(float)));
		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_x), &h_ff_flux_contribution_momentum_x, sizeof(float3)));
		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_y), &h_ff_flux_contribution_momentum_y, sizeof(float3)));
		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_z), &h_ff_flux_contribution_momentum_z, sizeof(float3)));

		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_density_energy), &h_ff_flux_contribution_density_energy, sizeof(float3)));

	}
	int nel;
	int nelr;

	// read in domain geometry
	float *areas;
	int *elements_surrounding_elements;
	float *normals;
	{
		std::ifstream file(data_file_name);

		file >> nel;
		nelr = block_length * ((nel / block_length) + std::min(1, nel % block_length));

		areas = new float[nelr];
		elements_surrounding_elements = new int[nelr * NNB];
		normals = new float[NDIM * NNB * nelr];

		// read in data
		for (int i = 0; i < nel; i++)
		{
			file >> areas[i];
			for (int j = 0; j < NNB; j++)
			{
				file >> elements_surrounding_elements[i + j * nelr];
				if (elements_surrounding_elements[i + j * nelr] < 0)
					elements_surrounding_elements[i + j * nelr] = -1;
				elements_surrounding_elements[i + j * nelr]--; // it's coming in with Fortran numbering

				for (int k = 0; k < NDIM; k++)
				{
					file >> normals[i + (j + k * NNB) * nelr];
					normals[i + (j + k * NNB) * nelr] = -normals[i + (j + k * NNB) * nelr];
				}
			}
		}

		// fill in remaining data
		int last = nel - 1;
		for (int i = nel; i < nelr; i++)
		{
			areas[i] = areas[last];
			for (int j = 0; j < NNB; j++)
			{
				// duplicate the last element
				elements_surrounding_elements[i + j * nelr] = elements_surrounding_elements[last + j * nelr];
				for (int k = 0; k < NDIM; k++)
					normals[i + (j + k * NNB) * nelr] = normals[last + (j + k * NNB) * nelr];
			}
		}
	}

	// Create arrays and set initial conditions
	float *variables = alloc<float>(nelr * NVAR);
	float *d_variables;
	gpuErrchk(hipMalloc((void **)&d_variables, sizeof(float) * nelr * NVAR));
	initialize_variables(nelr, d_variables);

	float *d_old_variables;
	gpuErrchk(hipMalloc((void **)&d_old_variables, sizeof(float) * nelr * NVAR));
	float *d_fluxes;
	gpuErrchk(hipMalloc((void **)&d_fluxes, sizeof(float) * nelr * NVAR));
	float *d_step_factors;
	gpuErrchk(hipMalloc((void **)&d_step_factors, sizeof(float) * nelr));


	float *d_areas, *d_normals;
	int *d_elements_surrounding_elements;
	gpuErrchk(hipMalloc((void **)&d_areas, sizeof(float)*nelr));
	gpuErrchk(hipMalloc((void **)&d_elements_surrounding_elements, sizeof(int)*nelr*NNB));
	gpuErrchk(hipMalloc((void **)&d_normals, sizeof(float)*NDIM * NNB * nelr));
	gpuErrchk(hipMemcpy(d_areas, areas, sizeof(float)*nelr, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_elements_surrounding_elements, elements_surrounding_elements, sizeof(float)*nelr*NNB, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_normals, normals, sizeof(float)*NDIM * NNB * nelr, hipMemcpyHostToDevice));

	//Chiamo il kernel per inizializzare le variabili
	//initialize_variables(nelr, old_variables, ff_variable);
	//initialize_variables(nelr, fluxes, ff_variable);

	// these need to be computed the first time in order to compute time step
	std::cout << "Starting..." << std::endl;

	// Begin iterations
	for (int i = 0; i < 2000; i++)
	{
		std::cout << i << "/" << iterations << std::endl;
		{
		gpuErrchk(hipMemcpy(d_old_variables, d_variables, sizeof(float) * nelr * NVAR, hipMemcpyDeviceToDevice));


		// for the first iteration we compute the time step
		//compute_step_factor(nelr, variables, areas, step_factors);
		//dim3 gridDim(nelr / BLOCK_SIZE, nelr/BLOCK_SIZE), blockDim(BLOCK_SIZE, BLOCK_SIZE);
		int blockDim = (nelr + BLOCK_SIZE - 1) / BLOCK_SIZE;
		compute_step_factor<<<blockDim, BLOCK_SIZE>>>(nelr, d_variables, d_areas, d_step_factors);
		gpuErrchk(hipPeekAtLastError());

		for (int j = 0; j < RK; j++)
		{
			//compute_flux(nelr, elements_surrounding_elements, normals, variables, fluxes, ff_variable, ff_flux_contribution_momentum_x, ff_flux_contribution_momentum_y, ff_flux_contribution_momentum_z, ff_flux_contribution_density_energy);
			compute_flux<<<blockDim, BLOCK_SIZE>>>(nelr, d_elements_surrounding_elements, d_normals, d_variables, d_fluxes);
			gpuErrchk(hipPeekAtLastError());
			//time_step(j, nelr, old_variables, variables, step_factors, fluxes);
			time_step<<<blockDim, BLOCK_SIZE>>>(j, nelr, d_old_variables, d_variables, d_step_factors, d_fluxes);
			gpuErrchk(hipPeekAtLastError());
		}
		}
	}

	gpuErrchk(hipMemcpy(variables, d_variables, sizeof(float) * nelr * NVAR, hipMemcpyDeviceToHost));

	std::cout << "Saving solution..." << std::endl;
	dump(variables, nel, nelr);
	std::cout << "Saved solution..." << std::endl;

	std::cout << "Cleaning up..." << std::endl;
	dealloc<float>(areas);
	dealloc<int>(elements_surrounding_elements);
	dealloc<float>(normals);

	dealloc<float>(variables);

	hipFree(d_variables);
	hipFree(d_old_variables);
	hipFree(d_step_factors);
	hipFree(d_fluxes);
	hipFree(d_areas);
	hipFree(d_elements_surrounding_elements);
	hipFree(d_normals);

	std::cout << "Done..." << std::endl;

	return 0;
}
